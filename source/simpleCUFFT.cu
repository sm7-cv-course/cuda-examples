#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <complex>
#include <math.h>
#include <iostream>
#include <valarray>
#include <time.h>
#include <stdlib.h>
// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_complex.h>
#include ""
#include "../common/book.h"
#include "../common/cpu_bitmap.h"

#define BLOCKSIZE 8
#define Dim 8
const float PI = 3.141592653589793238460;
typedef std::complex<double> stdComplex;

/*****************/
/* CUDA MEMCHECK */
/*****************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %dn", hipGetErrorString(code), file, line);
        if (abort) { getchar(); exit(code); }
    }
}
/*******************/
/* Host functions  */
/*******************/
int iDivUp(int hostPtr, int b) { return ((hostPtr % b) != 0) ? (hostPtr / b + 1) : (hostPtr / b); }

/*void
complex_to_bitmap(unsigned char *bm, stdComplex data_in[Dim], unsigned int w, unsigned int h) {
	unsigned int count = 0;

	for(unsigned int i=0; i < h; ++i) {
	    for(unsigned int j=0; j < w; ++j) {
	    	bm[count] = sqrt(std::real(data_in[i][j]) * std::real(data_in[i][j]) + std::imag(data_in) * std::imag(data_in));
	    	++count;
	    }
	}
}*/
/*******************/
/* Device functions */
/*******************/
__device__ double arg(double real, double imag)
{
    return atan(imag / real);
}
__device__ double abs_d(double real, double imag)
{
    return sqrt(real*real+imag*imag);
}
__device__ double real_d(double abs, double phase)
{
    return (abs*cos(phase));
}
__device__ double imag_d(double abs, double phase)
{
    return (abs*sin(phase)); //      !
}
__device__ double sqrt_d(double arg)
{
    return sqrt(arg);
}
/*******************/
/* Kernel 2D FUNCTION */
/*******************/
__global__ void MyKernel(hipfftDoubleComplex *d_target, hipfftDoubleComplex *data, size_t d_pitch, int nx, bool flag)
{
    int ind_y = threadIdx.y + blockDim.y*blockIdx.y;
    int ind_x = threadIdx.x + blockDim.x*blockIdx.x;
    int idx = ind_y + ind_x*nx;
    // data[idx].x /= (Dim*Dim);
    double Norm = (Dim);
    // data[idx] = hipCmul(make_hipDoubleComplex(Norm, 0.0), data[idx]);
    // if (flag) // Проверяем, в какой плоскости мы находимся и какую амплитуду меняем   true - плоскость образов
    {
        data[idx].x = abs_d(data[idx].x, data[idx].y);
        data[idx].y = arg(data[idx].x, data[idx].y);      // Преобразовали к амплитудно-фазовому
        data[idx].y /= (Norm);                            // Нормируем на размерность матрицы
        data[idx].x = d_target[idx].x;                    // Заменили амплитуду
        data[idx].x = real_d(data[idx].x, data[idx].y);   // Преобразуем к a + ib
        data[idx].y = imag_d(data[idx].x, data[idx].y);
    }
        // else
        // {
        //data[idx].x /= (Dim*Dim);
        //data[idx].y /= (Norm);
        //data[idx].x = d_target[idx].x;
        // }
}

void
bitmap_init(CPUBitmap * bitmap) {
	stdComplex target[Dim][Dim];
    stdComplex test[Dim][Dim];

    for(unsigned int i = 0; i < Dim; ++i) {
    	for(unsigned int j = 0; j < Dim; ++j) {
    		test[i][j] = stdComplex(PI * i / (i + j + 1), PI * j / (i + j + 1));
    		target[i][j] = stdComplex(0, 0);
    	}
    }

    stdComplex ISO[Dim][Dim];  // Отладочный массив  // Буфер хранения образов после преобразования Фурье
    memcpy(ISO, test, Dim * Dim * sizeof(stdComplex)); // Скопируем исходный массив чтобы сделать преобразование

    // Создаём из массива масок и амплитуд начального распределения массив вида a + ib.
    for (size_t i = 0; i < Dim; i++)
    {
        for (size_t j = 0; j < Dim; j++)
            ISO[i][j] = std::polar(real((ISO[i][j])), imag(ISO[i][j]));
    }

    // Выводим полученное.
    std::cout << " Initial matrix" << std::endl;
    for (size_t i = 0; i < Dim; i++)
    {
        for (size_t j = 0; j < Dim; j++)
            std::cout << ISO[i][j];
        std::cout << std::endl;
    }

    unsigned nx_mem_size = sizeof(stdComplex) * Dim;         // Если делать 2D-копирование с разделением
    size_t d_pitch, h_pitch=nx_mem_size ;                 // Если делать 2D-копирование с разделением
    size_t size = Dim * Dim * sizeof(hipfftDoubleComplex); // Копирование без разделения
    hipfftDoubleComplex *dataInOutCuda;
    hipfftDoubleComplex *data;
    hipfftDoubleComplex *d_target;
    hipfftDoubleComplex *d_basic;
    //gpuErrchk(hipMallocPitch((void**)&dataInOutCuda, &d_pitch, nx_mem_size, Dim));
    //gpuErrchk(hipMallocPitch((void**) &data, &d_pitch, nx_mem_size, Dim));
    //gpuErrchk(hipMallocPitch((void**)&d_target, &d_pitch, nx_mem_size, Dim));
    //gpuErrchk(hipMallocPitch((void**)&d_basic, &d_pitch, nx_mem_size, Dim));
    //gpuErrchk(hipMemcpy2D(d_target, d_pitch, target, h_pitch, nx_mem_size, Dim, hipMemcpyHostToDevice));
    //gpuErrchk(hipMemcpy2D(dataInOutCuda,d_pitch,  ISO, h_pitch, nx_mem_size, Dim, hipMemcpyHostToDevice));
    //gpuErrchk(hipMemcpy2D(d_basic, d_pitch, test, h_pitch, nx_mem_size, Dim, hipMemcpyHostToDevice));
    hipMalloc((void**)&dataInOutCuda, size);
    hipMalloc((void**)&data, size);
    hipMalloc((void**)&d_target, size);
    hipMalloc((void**)&d_basic, size);
    hipMemcpy(dataInOutCuda, ISO, size, hipMemcpyHostToDevice);   // Копируем подбираемое распределение на карту
    hipMemcpy(d_target, target, size, hipMemcpyHostToDevice);     // Копируем целевое распределение  на карту
    hipMemcpy(d_basic, test, size, hipMemcpyHostToDevice);        // Копируем  входное распределение на карту
    // std::cout<<std::endl << "host : " << h_pitch << " device : " << d_pitch << std::endl; /// Отладка
    ///----------Device code

    dim3 Grd(iDivUp(Dim, BLOCKSIZE), iDivUp(Dim, BLOCKSIZE));
    dim3 Blk(BLOCKSIZE, BLOCKSIZE);
    // Выделяем поле прямых и обратных преобразований.
    hipfftHandle planFFT, planIFFT;
    hipfftPlan2d(&planFFT, Dim, Dim, HIPFFT_Z2Z);
    hipfftPlan2d(&planIFFT, Dim, Dim, HIPFFT_Z2Z);

    for(size_t i = 0; i < 1; i++)
    {
        hipfftExecZ2Z(planFFT, dataInOutCuda, data, HIPFFT_FORWARD); //  Прямое преобразование Фурье для входящего распределения
        hipDeviceSynchronize();
        MyKernel << <Grd, Blk >> > (d_target, data, d_pitch, Dim, true); // Ядро, заменяющее полученную после прямого преобразования матрицу матрицей искомого
        hipDeviceSynchronize();
        hipfftExecZ2Z(planIFFT, data, dataInOutCuda, HIPFFT_BACKWARD); // Обратное преобразование Фурье с заменёнными амплитудами
        hipDeviceSynchronize();
        MyKernel << <Grd, Blk >> > (d_basic, dataInOutCuda, d_pitch, Dim, false); // Ядро, заменяющее полученную после обратного преобразования матрицу
                                                                                    // входным
        hipDeviceSynchronize();
    }
    stdComplex Tester[Dim][Dim];
    hipMemcpy(Tester, dataInOutCuda, size, hipMemcpyDeviceToHost); // Читаем с GPU
    ///   --- end device code

    // Выводим полученное
    std::cout << std::endl << "Here's Your successes " << std::endl << std::endl;
    for(size_t i = 0; i < Dim; i++)
    {
        for(size_t j = 0; j < Dim; j++)
            std::cout << Tester[i][j];
        std::cout << std::endl;
    }

    // complex_to_bitmap(bitmap.get_ptr(), Tester);
    unsigned int count = 0;
	for(unsigned int i=0; i < Dim; ++i) {
	    for(unsigned int j=0; j < Dim; ++j) {
	    	bitmap->get_ptr()[count] = sqrt(std::real(Tester[i][j]) * std::real(Tester[i][j]) + std::imag(Tester[i][j]) * std::imag(Tester[i][j])) * 125;
	    	++count;
	    }
	}

    hipfftDestroy(planFFT);
    hipfftDestroy(planIFFT);

    hipFree(dataInOutCuda);
    hipFree(data);
}


int main() {
	// Device bitmap.
	unsigned char *dev_bitmap;

	// CPU bitmap.
	CPUBitmap bitmap(Dim, Dim, dev_bitmap);

	HANDLE_ERROR(hipMalloc((void**)&dev_bitmap,
	                          bitmap.image_size()));

	// Form bitmap.
	bitmap_init(&bitmap);


	// Show bitmap.
	bitmap.display_and_exit();

    getchar();

    return 0;
}


/*int main()
{
    const stdComplex test[Dim][Dim] =
    {
        stdComplex(1.0, PI / 13), stdComplex(1.0,PI / 11), stdComplex(1.0,PI / 4), stdComplex(1.0,PI / 6), stdComplex(1.0, PI), stdComplex(1.0,5 * PI / 4), stdComplex(1.0,7 * PI / 4), stdComplex(1.0,6 * PI / 4),
        stdComplex(1.0,20 * PI / 17), stdComplex(1.0,11 * PI / 5), stdComplex(1.0,PI / 4), stdComplex(1.0,PI / 6), stdComplex(1.0, PI), stdComplex(1.0,5 * PI / 4), stdComplex(1.0,7 * PI / 4), stdComplex(1.0,6 * PI / 4),
        stdComplex(1.0,PI / 2), stdComplex(1.0,PI / 3), stdComplex(1.0,PI / 4), stdComplex(1.0,PI / 6), stdComplex(1.0, PI), stdComplex(1.0,5 * PI / 4), stdComplex(1.0,7 * PI / 4), stdComplex(1.0,6 * PI / 4),
        stdComplex(1.0,PI / 2), stdComplex(1.0,PI / 3), stdComplex(1.0,PI / 4), stdComplex(1.0,PI / 6), stdComplex(1.0, PI), stdComplex(1.0,5 * PI / 4), stdComplex(1.0,7 * PI / 4), stdComplex(1.0,6 * PI / 4),
        stdComplex(1.0,PI / 2), stdComplex(1.0,PI / 3), stdComplex(1.0,PI / 4), stdComplex(1.0,PI / 6), stdComplex(1.0, PI), stdComplex(1.0,5 * PI / 4), stdComplex(1.0,7 * PI / 4), stdComplex(1.0,6 * PI / 4),
        stdComplex(1.0,PI / 2), stdComplex(1.0,PI / 3), stdComplex(1.0,PI / 4), stdComplex(1.0,PI / 6), stdComplex(1.0, PI), stdComplex(1.0,5 * PI / 4), stdComplex(1.0,7 * PI / 4), stdComplex(1.0,6 * PI / 4),
        stdComplex(1.0,PI / 2), stdComplex(1.0,PI / 3), stdComplex(1.0,PI / 4), stdComplex(1.0,PI / 6), stdComplex(1.0, PI), stdComplex(1.0,5 * PI / 4), stdComplex(1.0,7 * PI / 4), stdComplex(1.0,6 * PI / 4),
        stdComplex(1.0,PI / 2), stdComplex(1.0,PI / 3), stdComplex(1.0,PI / 4), stdComplex(1.0,PI / 6), stdComplex(1.0, PI), stdComplex(1.0,5 * PI / 4), stdComplex(1.0,7 * PI / 4), stdComplex(1.0,6 * PI / 4)
    };

    stdComplex target[Dim][Dim] =
    {
        stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0),
        stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0),
        stdComplex(0,0), stdComplex(0,0), stdComplex(0.4,0), stdComplex(0.4,0), stdComplex(0.4,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0),
        stdComplex(0,0), stdComplex(0,0), stdComplex(0.4,0), stdComplex(1,0), stdComplex(1,0), stdComplex(0.4,0), stdComplex(0,0), stdComplex(0,0),
        stdComplex(0,0), stdComplex(0,0), stdComplex(0.4,0), stdComplex(1,0), stdComplex(1,0), stdComplex(0.4,0), stdComplex(0,0), stdComplex(0,0),
        stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0.4,0), stdComplex(0.4,0), stdComplex(0.4,0), stdComplex(0,0), stdComplex(0,0),
        stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0),
        stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0)
    };

    stdComplex ISO[Dim][Dim];  // Отладочный массив  // Буфер хранения образов после преобразования Фурье
    memcpy(ISO, test, Dim * Dim * sizeof(stdComplex)); // Скопируем исходный массив чтобы сделать преобразование

    // Создаём из массива масок и амплитуд начального распределения массив вида a + ib.
    for (size_t i = 0; i < Dim; i++)
    {
        for (size_t j = 0; j < Dim; j++)
            ISO[i][j] = std::polar(real((ISO[i][j])), imag(ISO[i][j]));

    }

    // Выводим полученное.
    std::cout << " Initial matrix" << std::endl;
    for (size_t i = 0; i < Dim; i++)
    {
        for (size_t j = 0; j < Dim; j++)
            std::cout << ISO[i][j];
        std::cout << std::endl;
    }

    unsigned nx_mem_size = sizeof(stdComplex) * Dim;    // Если делать 2D-копирование с разделением
    size_t d_pitch, h_pitch=nx_mem_size ;           // Если делать 2D-копирование с разделением
    size_t size = Dim * Dim * sizeof(hipfftDoubleComplex); // Копирование без разделения
    hipfftDoubleComplex *dataInOutCuda;
    hipfftDoubleComplex *data;
    hipfftDoubleComplex * d_target;
    hipfftDoubleComplex * d_basic;
    //gpuErrchk(hipMallocPitch((void**)&dataInOutCuda, &d_pitch, nx_mem_size, Dim));
    //gpuErrchk(hipMallocPitch ((void**) &data, &d_pitch, nx_mem_size, Dim));
    //gpuErrchk(hipMallocPitch((void**)&d_target, &d_pitch, nx_mem_size, Dim));
    //gpuErrchk(hipMallocPitch((void**)&d_basic, &d_pitch, nx_mem_size, Dim));
    //gpuErrchk(hipMemcpy2D(d_target, d_pitch, target, h_pitch, nx_mem_size, Dim, hipMemcpyHostToDevice));
    //gpuErrchk(hipMemcpy2D(dataInOutCuda,d_pitch,  ISO, h_pitch, nx_mem_size, Dim, hipMemcpyHostToDevice));
    //gpuErrchk(hipMemcpy2D(d_basic, d_pitch, test, h_pitch, nx_mem_size, Dim, hipMemcpyHostToDevice));
    hipMalloc((void**)&dataInOutCuda, size);
    hipMalloc((void**)&data, size);
    hipMalloc((void**)&d_target, size);
    hipMalloc((void**)&d_basic, size);
    hipMemcpy(dataInOutCuda, ISO, size, hipMemcpyHostToDevice);   // Копируем подбираемое распределение на карту
    hipMemcpy(d_target, target, size, hipMemcpyHostToDevice);     // Копируем целевое распределение  на карту
    hipMemcpy(d_basic, test, size, hipMemcpyHostToDevice);        // Копируем  входное распределение на карту
    // std::cout<<std::endl << "host : " << h_pitch << " device : " << d_pitch << std::endl; /// Отладка
    ///----------Device code

    dim3 Grd(iDivUp(Dim, BLOCKSIZE), iDivUp(Dim, BLOCKSIZE));
    dim3 Blk(BLOCKSIZE, BLOCKSIZE);
    // Выделяем поле прямых и обратных преобразований.
    hipfftHandle planFFT, planIFFT;
    hipfftPlan2d(&planFFT, Dim , Dim  , HIPFFT_Z2Z);
    hipfftPlan2d(&planIFFT, Dim , Dim , HIPFFT_Z2Z);

    for (size_t i = 0; i < 100; i++)
    {
        hipfftExecZ2Z(planFFT, dataInOutCuda, data, HIPFFT_FORWARD); //  Прямое преобразование Фурье для входящего распределения
        hipDeviceSynchronize();
        MyKernel << <Grd, Blk >> > (d_target, data, d_pitch, Dim, true); // Ядро, заменяющее полученную после прямого преобразования матрицу матрицей искомого
        hipDeviceSynchronize();
        hipfftExecZ2Z(planIFFT, data, dataInOutCuda, HIPFFT_BACKWARD); // Обратное преобразование Фурье с заменёнными амплитудами
        hipDeviceSynchronize();
        MyKernel << <Grd, Blk >> > (d_basic, dataInOutCuda, d_pitch, Dim, false); // Ядро, заменяющее полученную после обратного преобразования матрицу
                                                                                    // входным
        hipDeviceSynchronize();
    }
    stdComplex Tester[Dim][Dim];
    hipMemcpy(Tester, dataInOutCuda, size, hipMemcpyDeviceToHost); // Читаем с GPU
    ///   --- end device code

    // Выводим полученное
    std::cout << std::endl << "Here's Your successes " << std::endl << std::endl;
    for (size_t i = 0; i < Dim; i++)
    {
        for (size_t j = 0; j < Dim; j++)
            std::cout << Tester[i][j];
        std::cout << std::endl;
    }

    hipfftDestroy(planFFT);
    hipfftDestroy(planIFFT);

    hipFree(dataInOutCuda);
    hipFree(data);

    //delete[] dataOut, dataIn;

    getchar();

    return 0;
}*/
