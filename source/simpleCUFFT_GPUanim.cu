#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <complex>
#include <math.h>
#include <iostream>
#include <valarray>
#include <time.h>
#include <stdlib.h>
// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_complex.h>
#include ""
#include "../common/book.h"
// #include "../common/cpu_bitmap.h"
#include "../common/gpu_anim.h"

#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED   0.25f

// these exist on the GPU side
texture<float>  texConstSrc;
texture<float>  texIn;
texture<float>  texOut;

hipfftDoubleComplex *dataInOutCuda;
hipfftDoubleComplex *data;
hipfftDoubleComplex *d_target;
hipfftDoubleComplex *d_basic;
hipfftHandle planFFT, planIFFT;


#define BLOCKSIZE 8
#define Dim 8
const float PI = 3.141592653589793238460;
typedef std::complex<double> stdComplex;

/*****************/
/* CUDA MEMCHECK */
/*****************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

// globals needed by the update routine
struct DataBlock {
    float           *dev_inSrc;
    float           *dev_outSrc;
    float           *dev_constSrc;

    hipEvent_t     start, stop;
    float           totalTime;
    float           frames;
};

inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %dn", hipGetErrorString(code), file, line);
        if (abort) { getchar(); exit(code); }
    }
}
/*******************/
/* Host functions  */
/*******************/
int iDivUp(int hostPtr, int b) { return ((hostPtr % b) != 0) ? (hostPtr / b + 1) : (hostPtr / b); }

/*void
complex_to_bitmap(unsigned char *bm, stdComplex data_in[Dim], unsigned int w, unsigned int h) {
	unsigned int count = 0;

	for(unsigned int i=0; i < h; ++i) {
	    for(unsigned int j=0; j < w; ++j) {
	    	bm[count] = sqrt(std::real(data_in[i][j]) * std::real(data_in[i][j]) + std::imag(data_in) * std::imag(data_in));
	    	++count;
	    }
	}
}*/
/*******************/
/* Device functions */
/*******************/
__device__ double arg(double real, double imag)
{
    return atan(imag / real);
}
__device__ double abs_d(double real, double imag)
{
    return sqrt(real*real+imag*imag);
}
__device__ double real_d(double abs, double phase)
{
    return (abs*cos(phase));
}
__device__ double imag_d(double abs, double phase)
{
    return (abs*sin(phase)); //      !
}
__device__ double sqrt_d(double arg)
{
    return sqrt(arg);
}
/*******************/
/* Kernel 2D FUNCTION */
/*******************/
__global__ void MyKernel(hipfftDoubleComplex *d_target, hipfftDoubleComplex *data, size_t d_pitch, int nx, bool flag)
{
    int ind_y = threadIdx.y + blockDim.y * blockIdx.y;
    int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    int idx = ind_y + ind_x * nx;
    double Norm = (Dim);

    {
        data[idx].x = abs_d(data[idx].x, data[idx].y);
        data[idx].y = arg(data[idx].x, data[idx].y);      // Преобразовали к амплитудно-фазовому
        data[idx].y /= (Norm);                            // Нормируем на размерность матрицы
        data[idx].x = d_target[idx].x;                    // Заменили амплитуду
        data[idx].x = real_d(data[idx].x, data[idx].y);   // Преобразуем к a + ib
        data[idx].y = imag_d(data[idx].x, data[idx].y);
    }
}

__global__ void complex_to_double( double *optr, hipfftDoubleComplex *d_in_complex ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    optr[offset] = abs_d(d_in_complex.x, d_in_complex.y);
}

// NOTE - texOffsetConstSrc could either be passed as a
// parameter to this function, or passed in __constant__ memory
// if we declared it as a global above, it would be
// a parameter here:
// __global__ void copy_const_kernel( float *iptr,
//                                    size_t texOffset )
__global__ void copy_const_kernel( float *iptr ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float c = tex1Dfetch(texConstSrc,offset);
    if (c != 0)
        iptr[offset] = c;
}

void
bitmap_init(CPUBitmap * bitmap) {
	stdComplex target[Dim][Dim];
    stdComplex test[Dim][Dim];

    for(unsigned int i = 0; i < Dim; ++i) {
    	for(unsigned int j = 0; j < Dim; ++j) {
    		test[i][j] = stdComplex(PI * i / (i + j + 1), PI * j / (i + j + 1));
    		target[i][j] = stdComplex(0, 0);
    	}
    }

    stdComplex ISO[Dim][Dim];  // Отладочный массив  // Буфер хранения образов после преобразования Фурье
    memcpy(ISO, test, Dim * Dim * sizeof(stdComplex)); // Скопируем исходный массив, чтобы сделать преобразование

    // Создаём из массива масок и амплитуд начального распределения массив вида a + ib.
    for (size_t i = 0; i < Dim; i++)
    {
        for (size_t j = 0; j < Dim; j++)
            ISO[i][j] = std::polar(real((ISO[i][j])), imag(ISO[i][j]));
    }

    // Выводим полученное.
    std::cout << " Initial matrix" << std::endl;
    for (size_t i = 0; i < Dim; i++)
    {
        for (size_t j = 0; j < Dim; j++)
            std::cout << ISO[i][j];
        std::cout << std::endl;
    }

    unsigned nx_mem_size = sizeof(stdComplex) * Dim;         // Если делать 2D-копирование с разделением
    size_t d_pitch, h_pitch=nx_mem_size ;                 // Если делать 2D-копирование с разделением
    size_t size = Dim * Dim * sizeof(hipfftDoubleComplex); // Копирование без разделения
    /*hipfftDoubleComplex *dataInOutCuda;
    hipfftDoubleComplex *data;
    hipfftDoubleComplex *d_target;
    hipfftDoubleComplex *d_basic;*/
    //gpuErrchk(hipMallocPitch((void**)&dataInOutCuda, &d_pitch, nx_mem_size, Dim));
    //gpuErrchk(hipMallocPitch((void**) &data, &d_pitch, nx_mem_size, Dim));
    //gpuErrchk(hipMallocPitch((void**)&d_target, &d_pitch, nx_mem_size, Dim));
    //gpuErrchk(hipMallocPitch((void**)&d_basic, &d_pitch, nx_mem_size, Dim));
    //gpuErrchk(hipMemcpy2D(d_target, d_pitch, target, h_pitch, nx_mem_size, Dim, hipMemcpyHostToDevice));
    //gpuErrchk(hipMemcpy2D(dataInOutCuda,d_pitch,  ISO, h_pitch, nx_mem_size, Dim, hipMemcpyHostToDevice));
    //gpuErrchk(hipMemcpy2D(d_basic, d_pitch, test, h_pitch, nx_mem_size, Dim, hipMemcpyHostToDevice));
    hipMalloc((void**)&dataInOutCuda, size);
    hipMalloc((void**)&data, size);
    hipMalloc((void**)&d_target, size);
    hipMalloc((void**)&d_basic, size);
    hipMemcpy(dataInOutCuda, ISO, size, hipMemcpyHostToDevice);   // Копируем подбираемое распределение на карту
    hipMemcpy(d_target, target, size, hipMemcpyHostToDevice);     // Копируем целевое распределение  на карту
    hipMemcpy(d_basic, test, size, hipMemcpyHostToDevice);        // Копируем  входное распределение на карту
    // std::cout<<std::endl << "host : " << h_pitch << " device : " << d_pitch << std::endl; /// Отладка
    ///----------Device code

    Dim3 Grd(iDivUp(Dim, BLOCKSIZE), iDivUp(Dim, BLOCKSIZE));
    Dim3 Blk(BLOCKSIZE, BLOCKSIZE);
    // Выделяем поле прямых и обратных преобразований.
    // hipfftHandle planFFT, planIFFT;
    hipfftPlan2d(&planFFT, Dim, Dim, HIPFFT_Z2Z);
    hipfftPlan2d(&planIFFT, Dim, Dim, HIPFFT_Z2Z);

    for(size_t i = 0; i < 100; i++)
    {
        hipfftExecZ2Z(planFFT, dataInOutCuda, data, HIPFFT_FORWARD); //  Прямое преобразование Фурье для входящего распределения
        hipDeviceSynchronize();
        MyKernel << <Grd, Blk >> > (d_target, data, d_pitch, Dim, true); // Ядро, заменяющее полученную после прямого преобразования матрицу матрицей искомого
        hipDeviceSynchronize();
        hipfftExecZ2Z(planIFFT, data, dataInOutCuda, HIPFFT_BACKWARD); // Обратное преобразование Фурье с заменёнными амплитудами
        hipDeviceSynchronize();
        MyKernel << <Grd, Blk >> > (d_basic, dataInOutCuda, d_pitch, Dim, false); // Ядро, заменяющее полученную после обратного преобразования матрицу
                                                                                    // входным
        hipDeviceSynchronize();
    }
    stdComplex Tester[Dim][Dim];
    hipMemcpy(Tester, dataInOutCuda, size, hipMemcpyDeviceToHost); // Читаем с GPU
    ///   --- end device code

    // Выводим полученное
    std::cout << std::endl << "Here's Your successes " << std::endl << std::endl;
    for(size_t i = 0; i < Dim; i++)
    {
        for(size_t j = 0; j < Dim; j++)
            std::cout << Tester[i][j];
        std::cout << std::endl;
    }

    unsigned int count = 0;
	for(unsigned int i=0; i < Dim; ++i) {
	    for(unsigned int j=0; j < Dim; ++j) {
	    	bitmap->get_ptr()[count] = sqrt(std::real(Tester[i][j]) * std::real(Tester[i][j]) + std::imag(Tester[i][j]) * std::imag(Tester[i][j])) * 255;
	    	++count;
	    }
	}

    hipfftDestroy(planFFT);
    hipfftDestroy(planIFFT);

    hipFree(dataInOutCuda);
    hipFree(data);
}

void
anim_gpu( uchar4* outputBitmap, DataBlock *d, int ticks ) {
    HANDLE_ERROR( hipEventRecord( d->start, 0 ) );
    dim3    blocks(DIM/16,DIM/16);
    dim3    threads(16,16);

    // since tex is global and bound, we have to use a flag to
    // select which is in/out per iteration
    volatile bool dstOut = true;
    for (int i=0; i<90; i++) {
        float   *in, *out;
        if (dstOut) {
            in  = d->dev_inSrc;
            out = d->dev_outSrc;
        } else {
            out = d->dev_inSrc;
            in  = d->dev_outSrc;
        }
        copy_const_kernel<<<blocks,threads>>>( in );
        blend_kernel<<<blocks,threads>>>( out, dstOut );
        dstOut = !dstOut;
    }
    float_to_color<<<blocks,threads>>>( outputBitmap,
                                        d->dev_inSrc );

    HANDLE_ERROR( hipEventRecord( d->stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( d->stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        d->start, d->stop ) );
    d->totalTime += elapsedTime;
    ++d->frames;
    printf( "Average Time per frame:  %3.1f ms\n",
            d->totalTime/d->frames  );
}

// clean up memory allocated on the GPU
void
anim_exit( DataBlock *d ) {
    HANDLE_ERROR( hipUnbindTexture( texIn ) );
    HANDLE_ERROR( hipUnbindTexture( texOut ) );

    HANDLE_ERROR( hipFree( d->dev_inSrc ) );
    HANDLE_ERROR( hipFree( d->dev_outSrc ) );

    HANDLE_ERROR( hipEventDestroy( d->start ) );
    HANDLE_ERROR( hipEventDestroy( d->stop ) );
}

int
main() {
	DataBlock   data;
	GPUAnimBitmap bitmap( Dim, Dim, &data );
	data.totalTime = 0;
	data.frames = 0;
	HANDLE_ERROR( hipEventCreate( &data.start ) );
	HANDLE_ERROR( hipEventCreate( &data.stop ) );

	int imageSize = bitmap.image_size();

    // assume float == 4 chars in size (ie rgba)
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_inSrc,
                              imageSize ) );
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_outSrc,
                              imageSize ) );

    HANDLE_ERROR( hipBindTexture( NULL, texIn,
                                   data.dev_inSrc,
                                   imageSize ) );

    HANDLE_ERROR( hipBindTexture( NULL, texOut,
                                   data.dev_outSrc,
                                   imageSize ) );

    float *temp = (float*)malloc( imageSize );


    HANDLE_ERROR( hipMemcpy( data.dev_inSrc, temp,
                              imageSize,
                              hipMemcpyHostToDevice ) );
    free( temp );

    bitmap.anim_and_exit( (void (*)(uchar4*,void*,int))anim_gpu,
                           (void (*)(void*))anim_exit );
}

int main() {
	// Device bitmap.
	unsigned char *dev_bitmap;

	// CPU bitmap.
	CPUBitmap bitmap(Dim, Dim, dev_bitmap);

	HANDLE_ERROR(hipMalloc((void**)&dev_bitmap,
	                          bitmap.image_size()));

	// Form bitmap.
	bitmap_init(&bitmap);


	// Show bitmap.
	bitmap.display_and_exit();

    getchar();

    return 0;
}


/*int main()
{
    const stdComplex test[Dim][Dim] =
    {
        stdComplex(1.0, PI / 13), stdComplex(1.0,PI / 11), stdComplex(1.0,PI / 4), stdComplex(1.0,PI / 6), stdComplex(1.0, PI), stdComplex(1.0,5 * PI / 4), stdComplex(1.0,7 * PI / 4), stdComplex(1.0,6 * PI / 4),
        stdComplex(1.0,20 * PI / 17), stdComplex(1.0,11 * PI / 5), stdComplex(1.0,PI / 4), stdComplex(1.0,PI / 6), stdComplex(1.0, PI), stdComplex(1.0,5 * PI / 4), stdComplex(1.0,7 * PI / 4), stdComplex(1.0,6 * PI / 4),
        stdComplex(1.0,PI / 2), stdComplex(1.0,PI / 3), stdComplex(1.0,PI / 4), stdComplex(1.0,PI / 6), stdComplex(1.0, PI), stdComplex(1.0,5 * PI / 4), stdComplex(1.0,7 * PI / 4), stdComplex(1.0,6 * PI / 4),
        stdComplex(1.0,PI / 2), stdComplex(1.0,PI / 3), stdComplex(1.0,PI / 4), stdComplex(1.0,PI / 6), stdComplex(1.0, PI), stdComplex(1.0,5 * PI / 4), stdComplex(1.0,7 * PI / 4), stdComplex(1.0,6 * PI / 4),
        stdComplex(1.0,PI / 2), stdComplex(1.0,PI / 3), stdComplex(1.0,PI / 4), stdComplex(1.0,PI / 6), stdComplex(1.0, PI), stdComplex(1.0,5 * PI / 4), stdComplex(1.0,7 * PI / 4), stdComplex(1.0,6 * PI / 4),
        stdComplex(1.0,PI / 2), stdComplex(1.0,PI / 3), stdComplex(1.0,PI / 4), stdComplex(1.0,PI / 6), stdComplex(1.0, PI), stdComplex(1.0,5 * PI / 4), stdComplex(1.0,7 * PI / 4), stdComplex(1.0,6 * PI / 4),
        stdComplex(1.0,PI / 2), stdComplex(1.0,PI / 3), stdComplex(1.0,PI / 4), stdComplex(1.0,PI / 6), stdComplex(1.0, PI), stdComplex(1.0,5 * PI / 4), stdComplex(1.0,7 * PI / 4), stdComplex(1.0,6 * PI / 4),
        stdComplex(1.0,PI / 2), stdComplex(1.0,PI / 3), stdComplex(1.0,PI / 4), stdComplex(1.0,PI / 6), stdComplex(1.0, PI), stdComplex(1.0,5 * PI / 4), stdComplex(1.0,7 * PI / 4), stdComplex(1.0,6 * PI / 4)
    };

    stdComplex target[Dim][Dim] =
    {
        stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0),
        stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0),
        stdComplex(0,0), stdComplex(0,0), stdComplex(0.4,0), stdComplex(0.4,0), stdComplex(0.4,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0),
        stdComplex(0,0), stdComplex(0,0), stdComplex(0.4,0), stdComplex(1,0), stdComplex(1,0), stdComplex(0.4,0), stdComplex(0,0), stdComplex(0,0),
        stdComplex(0,0), stdComplex(0,0), stdComplex(0.4,0), stdComplex(1,0), stdComplex(1,0), stdComplex(0.4,0), stdComplex(0,0), stdComplex(0,0),
        stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0.4,0), stdComplex(0.4,0), stdComplex(0.4,0), stdComplex(0,0), stdComplex(0,0),
        stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0),
        stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0), stdComplex(0,0)
    };

    stdComplex ISO[Dim][Dim];  // Отладочный массив  // Буфер хранения образов после преобразования Фурье
    memcpy(ISO, test, Dim * Dim * sizeof(stdComplex)); // Скопируем исходный массив чтобы сделать преобразование

    // Создаём из массива масок и амплитуд начального распределения массив вида a + ib.
    for (size_t i = 0; i < Dim; i++)
    {
        for (size_t j = 0; j < Dim; j++)
            ISO[i][j] = std::polar(real((ISO[i][j])), imag(ISO[i][j]));

    }

    // Выводим полученное.
    std::cout << " Initial matrix" << std::endl;
    for (size_t i = 0; i < Dim; i++)
    {
        for (size_t j = 0; j < Dim; j++)
            std::cout << ISO[i][j];
        std::cout << std::endl;
    }

    unsigned nx_mem_size = sizeof(stdComplex) * Dim;    // Если делать 2D-копирование с разделением
    size_t d_pitch, h_pitch=nx_mem_size ;           // Если делать 2D-копирование с разделением
    size_t size = Dim * Dim * sizeof(hipfftDoubleComplex); // Копирование без разделения
    hipfftDoubleComplex *dataInOutCuda;
    hipfftDoubleComplex *data;
    hipfftDoubleComplex * d_target;
    hipfftDoubleComplex * d_basic;
    //gpuErrchk(hipMallocPitch((void**)&dataInOutCuda, &d_pitch, nx_mem_size, Dim));
    //gpuErrchk(hipMallocPitch ((void**) &data, &d_pitch, nx_mem_size, Dim));
    //gpuErrchk(hipMallocPitch((void**)&d_target, &d_pitch, nx_mem_size, Dim));
    //gpuErrchk(hipMallocPitch((void**)&d_basic, &d_pitch, nx_mem_size, Dim));
    //gpuErrchk(hipMemcpy2D(d_target, d_pitch, target, h_pitch, nx_mem_size, Dim, hipMemcpyHostToDevice));
    //gpuErrchk(hipMemcpy2D(dataInOutCuda,d_pitch,  ISO, h_pitch, nx_mem_size, Dim, hipMemcpyHostToDevice));
    //gpuErrchk(hipMemcpy2D(d_basic, d_pitch, test, h_pitch, nx_mem_size, Dim, hipMemcpyHostToDevice));
    hipMalloc((void**)&dataInOutCuda, size);
    hipMalloc((void**)&data, size);
    hipMalloc((void**)&d_target, size);
    hipMalloc((void**)&d_basic, size);
    hipMemcpy(dataInOutCuda, ISO, size, hipMemcpyHostToDevice);   // Копируем подбираемое распределение на карту
    hipMemcpy(d_target, target, size, hipMemcpyHostToDevice);     // Копируем целевое распределение  на карту
    hipMemcpy(d_basic, test, size, hipMemcpyHostToDevice);        // Копируем  входное распределение на карту
    // std::cout<<std::endl << "host : " << h_pitch << " device : " << d_pitch << std::endl; /// Отладка
    ///----------Device code

    Dim3 Grd(iDivUp(Dim, BLOCKSIZE), iDivUp(Dim, BLOCKSIZE));
    Dim3 Blk(BLOCKSIZE, BLOCKSIZE);
    // Выделяем поле прямых и обратных преобразований.
    hipfftHandle planFFT, planIFFT;
    hipfftPlan2d(&planFFT, Dim , Dim  , HIPFFT_Z2Z);
    hipfftPlan2d(&planIFFT, Dim , Dim , HIPFFT_Z2Z);

    for (size_t i = 0; i < 100; i++)
    {
        hipfftExecZ2Z(planFFT, dataInOutCuda, data, HIPFFT_FORWARD); //  Прямое преобразование Фурье для входящего распределения
        hipDeviceSynchronize();
        MyKernel << <Grd, Blk >> > (d_target, data, d_pitch, Dim, true); // Ядро, заменяющее полученную после прямого преобразования матрицу матрицей искомого
        hipDeviceSynchronize();
        hipfftExecZ2Z(planIFFT, data, dataInOutCuda, HIPFFT_BACKWARD); // Обратное преобразование Фурье с заменёнными амплитудами
        hipDeviceSynchronize();
        MyKernel << <Grd, Blk >> > (d_basic, dataInOutCuda, d_pitch, Dim, false); // Ядро, заменяющее полученную после обратного преобразования матрицу
                                                                                    // входным
        hipDeviceSynchronize();
    }
    stdComplex Tester[Dim][Dim];
    hipMemcpy(Tester, dataInOutCuda, size, hipMemcpyDeviceToHost); // Читаем с GPU
    ///   --- end device code

    // Выводим полученное
    std::cout << std::endl << "Here's Your successes " << std::endl << std::endl;
    for (size_t i = 0; i < Dim; i++)
    {
        for (size_t j = 0; j < Dim; j++)
            std::cout << Tester[i][j];
        std::cout << std::endl;
    }

    hipfftDestroy(planFFT);
    hipfftDestroy(planIFFT);

    hipFree(dataInOutCuda);
    hipFree(data);

    //delete[] dataOut, dataIn;

    getchar();

    return 0;
}*/
